#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------*/
/*  CP2K: A general program to perform molecular dynamics simulations         */
/*  Copyright 2000-2025 CP2K developers group <https://cp2k.org>              */
/*                                                                            */
/*  SPDX-License-Identifier: BSD-3-Clause                                     */
/*----------------------------------------------------------------------------*/

#include "../../offload/offload_runtime.h"
#if defined(__OFFLOAD) && !defined(__NO_OFFLOAD_GRID)

#include <algorithm>
#include <assert.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define GRID_DO_COLLOCATE 0
#include "../common/grid_common.h"
#include "grid_gpu_collint.h"
#include "grid_gpu_integrate.h"

// This has to be included after grid_gpu_collint.h
#include "../common/grid_process_vab.h"

#if defined(_OMP_H)
#error "OpenMP should not be used in .cu files to accommodate HIP."
#endif

// Teen registers are sufficient to integrate lp <= 2 with a single grid sweep.
#define GRID_N_CXYZ_REGISTERS 10

/*******************************************************************************
 * \brief Add value to designated register without using dynamic indexing.
 *        Otherwise the array would be stored in local memory, which is slower.
 * https://developer.nvidia.com/blog/fast-dynamic-indexing-private-arrays-cuda
 * \author Ole Schuett
 ******************************************************************************/
__device__ static inline void
add_to_register(const double value, const int index, cxyz_store *store) {
  switch (index) {
  case 0:
    store->regs[0] += value;
    break;
  case 1:
    store->regs[1] += value;
    break;
  case 2:
    store->regs[2] += value;
    break;
  case 3:
    store->regs[3] += value;
    break;
  case 4:
    store->regs[4] += value;
    break;
  case 5:
    store->regs[5] += value;
    break;
  case 6:
    store->regs[6] += value;
    break;
  case 7:
    store->regs[7] += value;
    break;
  case 8:
    store->regs[8] += value;
    break;
  case 9:
    store->regs[9] += value;
    break;
  }
}

/*******************************************************************************
 * \brief Integrate a single grid point with distance d{xyz} from center.
 * \author Ole Schuett
 ******************************************************************************/
__device__ static void gridpoint_to_cxyz(const double dx, const double dy,
                                         const double dz, const double zetp,
                                         const int lp, const double *gridpoint,
                                         cxyz_store *store) {

  // Squared distance of point from center.
  const double r2 = dx * dx + dy * dy + dz * dz;
  const double gaussian = exp(-zetp * r2);

  // Loading throught read-only cache reduces register usage for some reason.
  const double prefactor = __ldg(gridpoint) * gaussian;

  // Manually unrolled loops based on terms in coset_inv.
  if (store->offset == 0) {
    store->regs[0] += prefactor;
    if (lp >= 1) {
      store->regs[1] += prefactor * dx;
      store->regs[2] += prefactor * dy;
      store->regs[3] += prefactor * dz;
      if (lp >= 2) {
        store->regs[4] += prefactor * dx * dx;
        store->regs[5] += prefactor * dx * dy;
        store->regs[6] += prefactor * dx * dz;
        store->regs[7] += prefactor * dy * dy;
        store->regs[8] += prefactor * dy * dz;
        store->regs[9] += prefactor * dz * dz;
      }
    }

  } else if (store->offset == 10) {
    store->regs[0] += prefactor * dx * dx * dx;
    store->regs[1] += prefactor * dx * dx * dy;
    store->regs[2] += prefactor * dx * dx * dz;
    store->regs[3] += prefactor * dx * dy * dy;
    store->regs[4] += prefactor * dx * dy * dz;
    store->regs[5] += prefactor * dx * dz * dz;
    store->regs[6] += prefactor * dy * dy * dy;
    store->regs[7] += prefactor * dy * dy * dz;
    store->regs[8] += prefactor * dy * dz * dz;
    store->regs[9] += prefactor * dz * dz * dz;

  } else if (store->offset == 20) {
    store->regs[0] += prefactor * dx * dx * dx * dx;
    store->regs[1] += prefactor * dx * dx * dx * dy;
    store->regs[2] += prefactor * dx * dx * dx * dz;
    store->regs[3] += prefactor * dx * dx * dy * dy;
    store->regs[4] += prefactor * dx * dx * dy * dz;
    store->regs[5] += prefactor * dx * dx * dz * dz;
    store->regs[6] += prefactor * dx * dy * dy * dy;
    store->regs[7] += prefactor * dx * dy * dy * dz;
    store->regs[8] += prefactor * dx * dy * dz * dz;
    store->regs[9] += prefactor * dx * dz * dz * dz;

  } else if (store->offset == 30) {
    store->regs[0] += prefactor * dy * dy * dy * dy;
    store->regs[1] += prefactor * dy * dy * dy * dz;
    store->regs[2] += prefactor * dy * dy * dz * dz;
    store->regs[3] += prefactor * dy * dz * dz * dz;
    store->regs[4] += prefactor * dz * dz * dz * dz;
    if (lp >= 5) {
      store->regs[5] += prefactor * dx * dx * dx * dx * dx;
      store->regs[6] += prefactor * dx * dx * dx * dx * dy;
      store->regs[7] += prefactor * dx * dx * dx * dx * dz;
      store->regs[8] += prefactor * dx * dx * dx * dy * dy;
      store->regs[9] += prefactor * dx * dx * dx * dy * dz;
    }

  } else if (store->offset == 40) {
    store->regs[0] += prefactor * dx * dx * dx * dz * dz;
    store->regs[1] += prefactor * dx * dx * dy * dy * dy;
    store->regs[2] += prefactor * dx * dx * dy * dy * dz;
    store->regs[3] += prefactor * dx * dx * dy * dz * dz;
    store->regs[4] += prefactor * dx * dx * dz * dz * dz;
    store->regs[5] += prefactor * dx * dy * dy * dy * dy;
    store->regs[6] += prefactor * dx * dy * dy * dy * dz;
    store->regs[7] += prefactor * dx * dy * dy * dz * dz;
    store->regs[8] += prefactor * dx * dy * dz * dz * dz;
    store->regs[9] += prefactor * dx * dz * dz * dz * dz;

  } else if (store->offset == 50) {
    store->regs[0] += prefactor * dy * dy * dy * dy * dy;
    store->regs[1] += prefactor * dy * dy * dy * dy * dz;
    store->regs[2] += prefactor * dy * dy * dy * dz * dz;
    store->regs[3] += prefactor * dy * dy * dz * dz * dz;
    store->regs[4] += prefactor * dy * dz * dz * dz * dz;
    store->regs[5] += prefactor * dz * dz * dz * dz * dz;
    if (lp >= 6) {
      store->regs[6] += prefactor * dx * dx * dx * dx * dx * dx;
      store->regs[7] += prefactor * dx * dx * dx * dx * dx * dy;
      store->regs[8] += prefactor * dx * dx * dx * dx * dx * dz;
      store->regs[9] += prefactor * dx * dx * dx * dx * dy * dy;
    }

  } else if (store->offset == 60) {
    store->regs[0] += prefactor * dx * dx * dx * dx * dy * dz;
    store->regs[1] += prefactor * dx * dx * dx * dx * dz * dz;
    store->regs[2] += prefactor * dx * dx * dx * dy * dy * dy;
    store->regs[3] += prefactor * dx * dx * dx * dy * dy * dz;
    store->regs[4] += prefactor * dx * dx * dx * dy * dz * dz;
    store->regs[5] += prefactor * dx * dx * dx * dz * dz * dz;
    store->regs[6] += prefactor * dx * dx * dy * dy * dy * dy;
    store->regs[7] += prefactor * dx * dx * dy * dy * dy * dz;
    store->regs[8] += prefactor * dx * dx * dy * dy * dz * dz;
    store->regs[9] += prefactor * dx * dx * dy * dz * dz * dz;

  } else if (store->offset == 70) {
    store->regs[0] += prefactor * dx * dx * dz * dz * dz * dz;
    store->regs[1] += prefactor * dx * dy * dy * dy * dy * dy;
    store->regs[2] += prefactor * dx * dy * dy * dy * dy * dz;
    store->regs[3] += prefactor * dx * dy * dy * dy * dz * dz;
    store->regs[4] += prefactor * dx * dy * dy * dz * dz * dz;
    store->regs[5] += prefactor * dx * dy * dz * dz * dz * dz;
    store->regs[6] += prefactor * dx * dz * dz * dz * dz * dz;
    store->regs[7] += prefactor * dy * dy * dy * dy * dy * dy;
    store->regs[8] += prefactor * dy * dy * dy * dy * dy * dz;
    store->regs[9] += prefactor * dy * dy * dy * dy * dz * dz;

  } else if (store->offset == 80) {
    store->regs[0] += prefactor * dy * dy * dy * dz * dz * dz;
    store->regs[1] += prefactor * dy * dy * dz * dz * dz * dz;
    store->regs[2] += prefactor * dy * dz * dz * dz * dz * dz;
    store->regs[3] += prefactor * dz * dz * dz * dz * dz * dz;
    if (lp >= 7) {
      store->regs[4] += prefactor * dx * dx * dx * dx * dx * dx * dx;
      store->regs[5] += prefactor * dx * dx * dx * dx * dx * dx * dy;
      store->regs[6] += prefactor * dx * dx * dx * dx * dx * dx * dz;
      store->regs[7] += prefactor * dx * dx * dx * dx * dx * dy * dy;
      store->regs[8] += prefactor * dx * dx * dx * dx * dx * dy * dz;
      store->regs[9] += prefactor * dx * dx * dx * dx * dx * dz * dz;
    }

    // Handle higher offsets, ie. values of lp.
  } else {
    for (int i = 0; i < GRID_N_CXYZ_REGISTERS; i++) {
      double val = prefactor;
      const orbital a = coset_inv[i + store->offset];
      for (int j = 0; j < a.l[0]; j++) {
        val *= dx;
      }
      for (int j = 0; j < a.l[1]; j++) {
        val *= dy;
      }
      for (int j = 0; j < a.l[2]; j++) {
        val *= dz;
      }
      add_to_register(val, i, store);
    }
  }
}

/*******************************************************************************
 * \brief Integrates the grid into coefficients C_xyz.
 * \author Ole Schuett
 ******************************************************************************/
__device__ static void grid_to_cxyz(const kernel_params *params,
                                    const smem_task *task, const double *grid,
                                    double *cxyz) {

  // Atomics adds on shared memory are pretty slow. Hence, the coeffients are
  // accumulated in registers while looping over the grid points.
  // For larger values of lp we need to do multiple sweeps over the grid.
  // Due to the higher register usage and the multiple sweeps,
  // the integrate kernel runs about 70% slower than the collocate kernel.
  for (int offset = 0; offset < ncoset(task->lp);
       offset += GRID_N_CXYZ_REGISTERS) {

    double cxyz_regs[GRID_N_CXYZ_REGISTERS] = {0.0};
    cxyz_store store = {.regs = cxyz_regs, .offset = offset};

    if (task->use_orthorhombic_kernel) {
      ortho_cxyz_to_grid(params, task, &store, grid);
    } else {
      general_cxyz_to_grid(params, task, &store, grid);
    }

    // Add register values to coefficients stored in shared memory.
#pragma unroll // avoid dynamic indexing of registers
    for (int i = 0; i < GRID_N_CXYZ_REGISTERS; i++) {
      if (i + offset < ncoset(task->lp)) {
        atomicAddDouble(&cxyz[i + offset], cxyz_regs[i]);
      }
    }
  }
  __syncthreads(); // because of concurrent writes to cxyz
}

/*******************************************************************************
 * \brief Contracts the subblock, going from cartesian harmonics to spherical.
 * \author Ole Schuett
 ******************************************************************************/
template <bool COMPUTE_TAU>
__device__ static void store_hab(const smem_task *task, const cab_store *cab) {

  // The spherical index runs over angular momentum and then over contractions.
  // The carthesian index runs over exponents and then over angular momentum.

  // This is a double matrix product. Since the block can be quite large the
  // two products are fused to conserve shared memory.
  for (int i = threadIdx.x; i < task->nsgf_setb; i += blockDim.x) {
    for (int j = threadIdx.y; j < task->nsgf_seta; j += blockDim.y) {
      double block_val = 0.0;
      const int jco_start = ncoset(task->lb_min_basis - 1) + threadIdx.z;
      const int jco_end = ncoset(task->lb_max_basis);
      for (int jco = jco_start; jco < jco_end; jco += blockDim.z) {
        const orbital b = coset_inv[jco];
        const double sphib = task->sphib[i * task->maxcob + jco];
        const int ico_start = ncoset(task->la_min_basis - 1);
        const int ico_end = ncoset(task->la_max_basis);
        for (int ico = ico_start; ico < ico_end; ico++) {
          const orbital a = coset_inv[ico];
          const double hab =
              get_hab(a, b, task->zeta, task->zetb, cab, COMPUTE_TAU);
          const double sphia = task->sphia[j * task->maxcoa + ico];
          block_val += hab * sphia * sphib;
        }
      }
      if (task->block_transposed) {
        atomicAddDouble(&task->hab_block[j * task->nsgfb + i], block_val);
      } else {
        atomicAddDouble(&task->hab_block[i * task->nsgfa + j], block_val);
      }
    }
  }
  __syncthreads(); // Not needed, but coalesced threads are nice.
}

/*******************************************************************************
 * \brief Adds contributions from cab to forces and virial.
 * \author Ole Schuett
 ******************************************************************************/
template <bool COMPUTE_TAU>
__device__ static void store_forces_and_virial(const kernel_params *params,
                                               const smem_task *task,
                                               const cab_store *cab) {

  for (int i = threadIdx.x; i < task->nsgf_setb; i += blockDim.x) {
    for (int j = threadIdx.y; j < task->nsgf_seta; j += blockDim.y) {
      double block_val;
      if (task->block_transposed) {
        block_val = task->pab_block[j * task->nsgfb + i] * task->off_diag_twice;
      } else {
        block_val = task->pab_block[i * task->nsgfa + j] * task->off_diag_twice;
      }
      const int jco_start = ncoset(task->lb_min_basis - 1) + threadIdx.z;
      const int jco_end = ncoset(task->lb_max_basis);
      for (int jco = jco_start; jco < jco_end; jco += blockDim.z) {
        const double sphib = task->sphib[i * task->maxcob + jco];
        const int ico_start = ncoset(task->la_min_basis - 1);
        const int ico_end = ncoset(task->la_max_basis);
        for (int ico = ico_start; ico < ico_end; ico++) {
          const double sphia = task->sphia[j * task->maxcoa + ico];
          const double pabval = block_val * sphia * sphib;
          const orbital b = coset_inv[jco];
          const orbital a = coset_inv[ico];
          for (int k = 0; k < 3; k++) {
            const double force_a =
                get_force_a(a, b, k, task->zeta, task->zetb, cab, COMPUTE_TAU);
            atomicAddDouble(&task->forces_a[k], force_a * pabval);
            const double force_b = get_force_b(a, b, k, task->zeta, task->zetb,
                                               task->rab, cab, COMPUTE_TAU);
            atomicAddDouble(&task->forces_b[k], force_b * pabval);
          }
          if (params->virial != NULL) {
            for (int k = 0; k < 3; k++) {
              for (int l = 0; l < 3; l++) {
                const double virial_a = get_virial_a(
                    a, b, k, l, task->zeta, task->zetb, cab, COMPUTE_TAU);
                const double virial_b =
                    get_virial_b(a, b, k, l, task->zeta, task->zetb, task->rab,
                                 cab, COMPUTE_TAU);
                const double virial = pabval * (virial_a + virial_b);
                atomicAddDouble(&params->virial[k * 3 + l], virial);
              }
            }
          }
        }
      }
    }
  }
  __syncthreads(); // Not needed, but coalesced threads are nice.
}

/*******************************************************************************
 * \brief Initializes the cxyz matrix with zeros.
 * \author Ole Schuett
 ******************************************************************************/
__device__ static void zero_cxyz(const smem_task *task, double *cxyz) {
  if (threadIdx.z == 0 && threadIdx.y == 0) {
    for (int i = threadIdx.x; i < ncoset(task->lp); i += blockDim.x) {
      cxyz[i] = 0.0;
    }
  }
  __syncthreads(); // because of concurrent writes to cxyz
}

/*******************************************************************************
 * \brief Cuda kernel for integrating all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
template <bool COMPUTE_TAU, bool CALCULATE_FORCES>
__device__ static void integrate_kernel(const kernel_params *params) {

  // Copy task from global to shared memory and precompute some stuff.
  __shared__ smem_task task;
  load_task(params, &task);

  // Check if radius is below the resolution of the grid.
  if (2.0 * task.radius < task.dh_max) {
    return; // nothing to do
  }

  // Allot dynamic shared memory.
  extern __shared__ double shared_memory[];
  double *smem_cab = &shared_memory[params->smem_cab_offset];
  double *smem_alpha = &shared_memory[params->smem_alpha_offset];
  double *smem_cxyz = &shared_memory[params->smem_cxyz_offset];

  // Allocate Cab from global memory if it does not fit into shared memory.
  cab_store cab = {.data = NULL, .n1 = task.n1};
  if (params->smem_cab_length < task.n1 * task.n2) {
    cab.data = malloc_cab(&task);
  } else {
    cab.data = smem_cab;
  }

  zero_cab(&cab, task.n1 * task.n2);
  compute_alpha(&task, smem_alpha);

  zero_cxyz(&task, smem_cxyz);
  grid_to_cxyz(params, &task, params->grid, smem_cxyz);
  cab_to_cxyz(&task, smem_alpha, &cab, smem_cxyz);

  store_hab<COMPUTE_TAU>(&task, &cab);
  if (CALCULATE_FORCES) {
    store_forces_and_virial<COMPUTE_TAU>(params, &task, &cab);
  }

  if (params->smem_cab_length < task.n1 * task.n2) {
    free_cab(cab.data);
  }
}

/*******************************************************************************
 * \brief Specialized Cuda kernel for compute_tau=false & calculate_forces=false
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void grid_integrate_density(const kernel_params params) {
  integrate_kernel<false, false>(&params);
}

/*******************************************************************************
 * \brief Specialized Cuda kernel for compute_tau=true & calculate_forces=false.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void grid_integrate_tau(const kernel_params params) {
  integrate_kernel<true, false>(&params);
}

/*******************************************************************************
 * \brief Specialized Cuda kernel for compute_tau=false & calculate_forces=true.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void
grid_integrate_density_forces(const kernel_params params) {
  integrate_kernel<false, true>(&params);
}

/*******************************************************************************
 * \brief Specialized Cuda kernel for compute_tau=true & calculate_forces=true.
 * \author Ole Schuett
 ******************************************************************************/
__global__ static void grid_integrate_tau_forces(const kernel_params params) {
  integrate_kernel<true, true>(&params);
}

/*******************************************************************************
 * \brief Launches the Cuda kernel that integrates all tasks of one grid level.
 * \author Ole Schuett
 ******************************************************************************/
void grid_gpu_integrate_one_grid_level(
    const grid_gpu_task_list *task_list, const int first_task,
    const int last_task, const bool compute_tau, const grid_gpu_layout *layout,
    const offloadStream_t stream, const double *pab_blocks_dev,
    const double *grid_dev, double *hab_blocks_dev, double *forces_dev,
    double *virial_dev, int *lp_diff) {

  // Compute max angular momentum.
  const bool calculate_forces = (forces_dev != NULL);
  const bool calculate_virial = (virial_dev != NULL);
  assert(!calculate_virial || calculate_forces);
  const process_ldiffs ldiffs =
      process_get_ldiffs(calculate_forces, calculate_virial, compute_tau);
  *lp_diff = ldiffs.la_max_diff + ldiffs.lb_max_diff; // for reporting stats
  const int la_max = task_list->lmax + ldiffs.la_max_diff;
  const int lb_max = task_list->lmax + ldiffs.lb_max_diff;
  const int lp_max = la_max + lb_max;

  const int ntasks = last_task - first_task + 1;
  if (ntasks == 0) {
    return; // Nothing to do and lp_diff already set.
  }

  init_constant_memory();

  // Small Cab blocks are stored in shared mem, larger ones in global memory.
  const int CAB_SMEM_LIMIT = ncoset(5) * ncoset(5); // = 56 * 56 = 3136

  // Compute required shared memory.
  const int alpha_len = 3 * (lb_max + 1) * (la_max + 1) * (lp_max + 1);
  const int cxyz_len = ncoset(lp_max);
  const int cab_len = imin(CAB_SMEM_LIMIT, ncoset(lb_max) * ncoset(la_max));
  const size_t smem_per_block =
      (alpha_len + cxyz_len + cab_len) * sizeof(double);

  // kernel parameters
  kernel_params params;
  params.smem_cab_length = cab_len;
  params.smem_cab_offset = 0;
  params.smem_alpha_offset = params.smem_cab_offset + cab_len;
  params.smem_cxyz_offset = params.smem_alpha_offset + alpha_len;
  params.first_task = first_task;
  params.grid = grid_dev;
  params.tasks = task_list->tasks_dev;
  params.pab_blocks = pab_blocks_dev;
  params.hab_blocks = hab_blocks_dev;
  params.forces = forces_dev;
  params.virial = virial_dev;
  params.la_min_diff = ldiffs.la_min_diff;
  params.lb_min_diff = ldiffs.lb_min_diff;
  params.la_max_diff = ldiffs.la_max_diff;
  params.lb_max_diff = ldiffs.lb_max_diff;
  memcpy(params.dh, layout->dh, 9 * sizeof(double));
  memcpy(params.dh_inv, layout->dh_inv, 9 * sizeof(double));
  memcpy(params.npts_global, layout->npts_global, 3 * sizeof(int));
  memcpy(params.npts_local, layout->npts_local, 3 * sizeof(int));
  memcpy(params.shift_local, layout->shift_local, 3 * sizeof(int));

  // Launch !
  const int nblocks = ntasks;
  const dim3 threads_per_block(4, 4, 4);

  if (!compute_tau && !calculate_forces) {
    grid_integrate_density<<<nblocks, threads_per_block, smem_per_block,
                             stream>>>(params);
  } else if (compute_tau && !calculate_forces) {
    grid_integrate_tau<<<nblocks, threads_per_block, smem_per_block, stream>>>(
        params);
  } else if (!compute_tau && calculate_forces) {
    grid_integrate_density_forces<<<nblocks, threads_per_block, smem_per_block,
                                    stream>>>(params);
  } else if (compute_tau && calculate_forces) {
    grid_integrate_tau_forces<<<nblocks, threads_per_block, smem_per_block,
                                stream>>>(params);
  }
  OFFLOAD_CHECK(offloadGetLastError());
}

#endif // defined(__OFFLOAD) && !defined(__NO_OFFLOAD_GRID)
// EOF
